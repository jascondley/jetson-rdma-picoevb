#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <errno.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/ioctl.h>
#include <sys/stat.h>
#include <sys/types.h>
#include "../kernel-module/picoevb-rdma-ioctl.h"

#define SURFACE_W  256
#define SURFACE_H  256
#define SURFACE_SIZE  (SURFACE_W * SURFACE_H)

#define OFFSET(x, y)  (((y) * SURFACE_W) + x)
#define DATA(x, y)  (((y & 0xffff) << 16) | ((x) & 0xffff))

extern "C" __global__ void fill_surface(uint32_t *output, uint32_t xor_val)
{
  unsigned int pos_x = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int pos_y = (blockIdx.y * blockDim.y) + threadIdx.y;

  output[OFFSET(pos_x, pos_y)] = DATA(pos_x, pos_y) ^ xor_val;
}

extern "C" __global__ void reorder_bytes(uint32_t* gpu_data)
{
  unsigned int pos_x = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int pos_y = (blockIdx.y * blockDim.y) + threadIdx.y;

  const uint32_t word = gpu_data[OFFSET(pos_x, pos_y)];
  const uint8_t b0 = word & 0xFF;
  const uint8_t b1 = (word >> 8) & 0xFF;
  const uint8_t b2 = (word >> 16) & 0xFF;
  const uint8_t b3 = (word >> 24) & 0xFF;

  gpu_data[OFFSET(pos_x, pos_y)] = (b0 << 24) | (b1 << 16) | (b2 << 8) | b3;
}

void HexDump(const uint8_t* bytes, size_t size)
{
  if (!size) return;

  const size_t bytes_per_line = 16;
  const size_t total_lines = ((size - 1) / bytes_per_line) + 1;

  for (size_t line = 0; line < total_lines; ++line) {
    const unsigned int offset = line * bytes_per_line;
    // Show the offset each line:
    printf("%08x", offset);

    // Hex bytes
    for (size_t i = offset; i < offset + bytes_per_line; ++i) {
      // Add a bit of space for visual clarity
      if (i % (bytes_per_line / 2) == 0)
        printf(" ");
      if (i < size)
        printf(" %02x", bytes[i]);
      else
        printf("   ");
    }

    // printable characters
    printf("  ");
    for (size_t i = offset; i < offset + bytes_per_line && i < size; ++i) {
      char c = bytes[i];
      const char first_printable = ' ';
      const char last_printable = '\x7e';
      if (c < first_printable || c > last_printable)
        printf(".");
      else
        printf("%c", c);
    }
    printf("\n");
    fflush(stdout);
  }
}

int main(int argc, char **argv)
{
  hipError_t ce;
  hipError_t cr;
  uint32_t* src_d;
  int fd, ret;
  unsigned int flag = 1;
  struct picoevb_rdma_pin_cuda pin_params_src;
  struct picoevb_rdma_h2c2h_dma dma_params;
  struct picoevb_rdma_unpin_cuda unpin_params_src;

  if (argc != 1) {
    fprintf(stderr, "usage: cuda-babe\n");
    return 1;
  }

  fd = open("/dev/picoevb", O_RDWR);
  if (fd < 0) {
    perror("open() failed");
    return 1;
  }

  ce = hipHostAlloc(&src_d, SURFACE_SIZE * sizeof(*src_d),
    hipHostMallocDefault);

  if (ce != hipSuccess) {
    fprintf(stderr, "Allocation of src_d failed: %d\n", ce);
    return 1;
  }

  cr = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,
    (hipDeviceptr_t)src_d);
  if (cr != hipSuccess) {
    fprintf(stderr, "hipPointerSetAttribute(src_d) failed: %d\n", cr);
    return 1;
  }

  pin_params_src.va = (__u64)src_d;
  pin_params_src.size = SURFACE_SIZE * sizeof(*src_d);
  ret = ioctl(fd, PICOEVB_IOC_PIN_CUDA, &pin_params_src);
  if (ret != 0) {
    fprintf(stderr, "ioctl(PIN_CUDA src) failed: ret=%d errno=%d\n", ret, errno);
    return 1;
  }

#if (SURFACE_W < 16) || (SURFACE_H < 16)
#error Grid and block sizes must be shrunk for small surfaces
#endif
#if (SURFACE_W & 15) || (SURFACE_H & 15)
#error Grid and block sizes are not a multiple of the surface size
#endif
  // Here's the missing piece. There needs to be some way to get the
  // memory from FPGA directly to the src_d that we've pinned. I'm going to
  // start by assuming that the memory has to be pinned to DMA to do it.
  dma_params.src = (__u64)src_d;
  dma_params.dst = 0;
  dma_params.len = SURFACE_SIZE * sizeof(*src_d);
  dma_params.flags = 0;
  ret = ioctl(fd, PICOEVB_IOC_C2H_DMA, &dma_params);
  if (ret != 0) {
    fprintf(stderr, "ioctl(DMA) failed: %d\n", ret);
    perror("ioctl() failed");
    return 1;
  }

  unpin_params_src.handle = pin_params_src.handle;
  ret = ioctl(fd, PICOEVB_IOC_UNPIN_CUDA, &unpin_params_src);
  if (ret != 0) {
    fprintf(stderr, "ioctl(UNPIN_CUDA src) failed: %d\n", ret);
    return 1;
  }

  dim3 dimGrid(SURFACE_W / 16, SURFACE_H / 16);
  dim3 dimBlock(16, 16);
  reorder_bytes<<<dimGrid, dimBlock>>>(src_d);

  ce = hipDeviceSynchronize();
  if (ce != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize() failed: %d\n", ce);
    return 1;
  }

  // If this works, it's because of some weird zero-copy logic.
  HexDump((uint8_t*)src_d, SURFACE_SIZE * sizeof(*src_d));

  ce = hipHostFree(src_d);

  if (ce != hipSuccess) {
    fprintf(stderr, "Free of src_d failed: %d\n", ce);
    return 1;
  }

  ret = close(fd);
  if (ret < 0) {
    perror("close() failed");
    return 1;
  }
}

